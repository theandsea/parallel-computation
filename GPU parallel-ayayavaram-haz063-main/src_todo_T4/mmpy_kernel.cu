#include "hip/hip_runtime.h"
// ;-*- mode: c;-*-
// Matrix multiply device code



#include <assert.h>
#include <math.h>
#include "../src/utils.h"
#include "../src/types.h"
#include "mytypes.h"
using namespace std;

#include <stdio.h>

extern __shared__ _FTYPE_ sharemem[];

#define stride_y 16 // keep strides x and y the same as bx and by in OPTIONS.TXT (block dimensions)
#define stride_x 32
#define mult_y TILEDIM_M/stride_y
#define mult_x TILEDIM_N/stride_x
#define mult_k_y TILEDIM_K/stride_y
#define mult_k_x TILEDIM_K/stride_x


#ifdef NAIVE
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {

    int I =  blockIdx.y*blockDim.y + threadIdx.y;
    int J =  blockIdx.x*blockDim.x + threadIdx.x;

    if((I < N) && (J < N)){
        _FTYPE_ _c = 0;
        for (unsigned int k = 0; k < N; k++) {
            _FTYPE_ a = A[I * N + k];
            _FTYPE_ b = B[k * N + J];
            _c += a * b;
        }
        C[I * N + J] = _c;
    }
}


#else
//You should be changing the kernel here for the non naive implementation.
__global__ void matMul(int N, _FTYPE_ *C, _FTYPE_ *A, _FTYPE_ *B) {
    
    //local shared storage
    // __shared__ _FTYPE_ As[TW][TW], Bs[TW][TW];

    int ty = threadIdx.y, tx = threadIdx.x;
    int by = blockIdx.y, bx = blockIdx.x;
    
    int I = by*TILEDIM_M + ty, J = bx*TILEDIM_N + tx;

    // 4 elements of Cs calculated together
    // this thread memory is also limited, but if just 0, it can ignore
    _FTYPE_ Cij[mult_y][mult_x] = {0.0f};

    _FTYPE_ * __restrict__ As = &sharemem[0];
    _FTYPE_ * __restrict__ Bs = &As[TILEDIM_M*TILEDIM_K];

    int t_K = TILEDIM_K, t_K0 = TILEDIM_K;


    // #pragma unroll // this unroll will wiredly decrease the performance
    // when used in inside loop, not difference
    for (int kk=0; kk<N; kk+=TILEDIM_K) {
        if (t_K > N-kk)  t_K0 = N-kk;

        // #pragma unroll
        for(int ii=0; ii<mult_y; ii++)
            for(int jj=0; jj<mult_k_x; jj++)
                // if is slow...do not diverge
                As[(ty+ii*stride_y)*t_K + tx + jj*stride_x] = (I+ii*stride_y<N && kk + tx + jj*stride_x<N)? A[((I+ii*stride_y)*N + kk + tx + jj*stride_x)]:0.0f ;

        //#pragma unroll
        for(int ii=0; ii<mult_k_y; ii++)
            for(int jj=0; jj<mult_x; jj++)
                Bs[(ty+ii*stride_y)*TILEDIM_N + tx + jj*stride_x] = (kk+ty+ii*stride_y <N && J+jj*stride_x <N) ? B[((kk+ty+ii*stride_y)*N + J+jj*stride_x)]:0.0f;

        __syncthreads();

        // #pragma unroll
        for (int k=0; k<t_K0; k++) 
            for(int ii=0; ii<mult_y; ii++)
                for(int jj=0; jj<mult_x; jj++)
                    Cij[ii][jj]+= As[(ty+ii*stride_y)*TILEDIM_K + k] * Bs[k*TILEDIM_N + tx + jj*stride_x];        
        __syncthreads(); // necessary otherwise As updated
    }

    // put it outside, or run out of resource, too much unrolled instructions...each thread only has 256 register as memory
    // #pragma unroll
    for(int ii=0; ii<mult_y; ii++)
        for(int jj=0; jj<mult_x; jj++)
            if (I+ii*stride_y<N && J+jj*stride_x<N) C[(I+ii*stride_y)*N + J+jj*stride_x] = Cij[ii][jj];
}

#endif
